/*
 * File: kernels.cu
 * License: Please see LICENSE file.
 * AccFFT: Massively Parallel FFT Library
 * Created by Amir Gholami on 06/04/2015
 * Email: contact@accfft.org
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

#define TPL_DECL(proto) proto(float) proto(double)
#define TCASE(real) template real testcase<real>(real X, real Y, real Z);
#define TCASE_GPU(real) template __device__ real testcase_gpu<real>(real X, real Y, real Z);
#define INIT_GPU(real) template void initialize_gpu(real *a, int*n, int * isize, int * istart);

template <typename real>
real testcase(real X, real Y, real Z) {
	real sigma = 4;
	real pi = M_PI;
	real analytic;
	analytic = std::exp(-sigma * ((X - pi) * (X - pi) + (Y - pi) * (Y - pi)
							+ (Z - pi) * (Z - pi)));
	if (analytic != analytic)
		analytic = 0;
	return analytic;
}

template <typename real>
__device__ real testcase_gpu(real X, real Y, real Z) {

	real sigma = 4;
	real pi = M_PI;
	real analytic;
	analytic = std::exp(-sigma * ((X - pi) * (X - pi) + (Y - pi) * (Y - pi)
							+ (Z - pi) * (Z - pi)));
	if (analytic != analytic)
		analytic = 0;
	return analytic;
}

template <typename real>
__global__ void initialize_gpu_kernel(real * a, int *n, int n2_, int* isize,
		int* istart) {
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int j = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int k = blockDim.z * blockIdx.z + threadIdx.z;

	if (i >= isize[0])
		return;
	if (j >= isize[1])
		return;
	if (k >= isize[2])
		return;

	{
		real pi = M_PI;
		real X, Y, Z;
		long int ptr;

		X = 2 * pi / n[0] * (i + istart[0]);
		Y = 2 * pi / n[1] * (j + istart[1]);
		Z = 2 * pi / n[2] * k;

		ptr = i * isize[1] * n2_ + j * n2_ + k;
		a[ptr] = testcase_gpu(X, Y, Z);
	}
	return;

} // end initialize_gpu_kernel

template <typename real>
void initialize_gpu(real *a, int*n, int * isize, int * istart) {

	int n2_ = (n[2] / 2 + 1) * 2; // Due to inplace padding

	// corresponding GPU sizes
	int * n_gpu, *isize_gpu, *istart_gpu;
	hipMalloc((void**) &n_gpu, 3 * sizeof(int));
	hipMalloc((void**) &isize_gpu, 3 * sizeof(int));
	hipMalloc((void**) &istart_gpu, 3 * sizeof(int));

	// Copy the sizes to GPU
	hipMemcpy(n_gpu, n, 3 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(isize_gpu, isize, 3 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(istart_gpu, istart, 3 * sizeof(int), hipMemcpyHostToDevice);

	int blocksInX = std::ceil(n[0] / 4.);
	int blocksInY = std::ceil(n[1] / 4.);
	int blocksInZ = std::ceil(n2_ / 4.);

	dim3 Dg(blocksInX, blocksInY, blocksInZ);
	dim3 Db(4, 4, 4);
	initialize_gpu_kernel<<<Dg, Db>>>(a,n_gpu,n2_,isize_gpu,istart_gpu);
	hipDeviceSynchronize();

	hipFree(n_gpu);
	hipFree(isize_gpu);
	hipFree(istart_gpu);
	return;
} // end initialize_gpu

TPL_DECL(TCASE)
TPL_DECL(TCASE_GPU)
TPL_DECL(INIT_GPU)

